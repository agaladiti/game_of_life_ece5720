#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define ITER 20
#define BILLION 1000000000

void write_output(int *mat, int m, int n, FILE *res)
{
  for (int i = 0; i < m; i++)
  {
    for (int j = 0; j < n; j++)
    {
      fprintf(res, "%d ", mat[i*n+j]);
    }
    fprintf(res, "\n");
  }
  for (int i = 0; i < 3; i++)
  {
    fprintf(res, "\n");
  }
}

__global__ void update_matrix(int *current, int *future, int m, int n)
{
  int x = blockIdx.y * blockDim.y + threadIdx.y;
  int y = blockIdx.x * blockDim.x + threadIdx.x;
  
  for (int i = 1; i < m - 1; i++)
  {
    for (int j = 1; j < n - 1; j++)
    {
      int aliveN = 0;
      for (int i = -1; i <= 1; i++)
      {
        for (int j = -1; j <= 1; j++)
        {
          aliveN += current[(x + i)*m + y + j];
        }
      }
      aliveN -= current[x*m + y];

      //if lonely it dies
      if (aliveN < 2 && current[x*m + y] == 1)
        future[x*m + y] = 0;
      //if overpopulated it dies
      else if (aliveN > 3 && current[x*m + y] == 1)
      {
        future[x*m + y] = 0;
      }
      // if repopulated it revives
      else if (aliveN == 3 && current[x*m + y] == 0)
        future[x*m + y] = 1;
      // else copy current to future
      else
      {
        future[x*m + y] = current[x*m + y];
      }
    }
  }
}

int main()
{
  int i, j;
  int m, n;
  int *dev_even, *dev_odd;

  FILE *res;
  res = fopen("output.txt", "w");

  m = n = 8;

  int *even = (int*) calloc(m * n *sizeof(int), sizeof(int));
  for (i = 1; i < m-1; i++)
  {
    for (j = 1; j < n - 1; j++)
    {
      even[i*m+j] = rand() % 2;
    }
  }
  int *odd = (int *) calloc(m * n *sizeof(int), sizeof(int));

  dim3 Block(m,n);
  dim3 Grid(1,1);



  for (int iter = 0; iter < ITER; iter++)
  {
    if (iter % 2 == 0)
    {
      hipMemcpy(dev_even,even,m*n*sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(dev_odd,odd,m*n*sizeof(int),hipMemcpyHostToDevice);
      update_matrix<<<Grid, Block>>>(dev_even,dev_odd,m,n);
      hipMemcpy(odd,dev_odd,m*n*sizeof(int),hipMemcpyDeviceToHost);
      write_output(odd, m, n, res);
    }
    if (iter % 2 == 1)
    {
      hipMemcpy(dev_even,even,m*n*sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(dev_odd,odd,m*n*sizeof(int),hipMemcpyHostToDevice);
      update_matrix<<<Grid, Block>>>(dev_odd,dev_even,m,n);
      hipMemcpy(even,dev_even,m*n*sizeof(int),hipMemcpyDeviceToHost);
      write_output(even, m, n, res);
    }
  }
  fclose(res);
  free(even);
  free(odd);
}
